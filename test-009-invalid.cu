#include "hip/hip_runtime.h"
// This file contains various type errors in the CU language.
// It is not a complete test.
function f(ref |float| x) { }
function g(|float| x) { }
function h(|float| x) returns |int| { return round x; } 

function main( ) returns |int|
{
    |int| a;      
    array of |int| data;  
    pointer to |int| p;

    |int| a;              // Duplicate identifier      
    |double| x;           // Unknown type name

    y = 42;               // Unknown identifier
    f(4.1);               // L-Value expected for ref param
    f(a);                 // Invalid argument type
    g(4.1);               // OK
    g(a);                 // OK
    g("Hello");           // Invalid argument type
    g();                  // Wrong number of arguments
    g(4.2, 2.2);          // Wrong number of arguments
    a = h;                // Function name used incorrectly
    a = g(4.1);           // OK
    data = (array of |int| is 1, 2.2 );// Wrong type in array constant
    p = (new |int| is 2.2);            // Wrong type in new pointer
    if (a and true)then write 0; fi    // Expected boolean
    if (a > "Hello") then write 1; fi  // Expected same type
    a = 4 - "Hello";                   // Type error
    a = *a;                            // Type error
    read 42;                           // Must be addressable
    write data;                        // Must be basic type
    if (a) then write 2; fi            // Expected boolean
    return 4.2;                        // Wrong return type
}
