|int| x;
|int| y is initially 42;

function f(|int| x) returns |int|
{
    return x*x*x;
}

function test_syntax 
(|string| s, |int| x, |float| y, |bool| z)
{
    pointer to |int| p;
    array of |int| n;
    |int| i;

    x = 42;
    y = 42.8e+9;
    s = "Hello, World!";
    z = true;
    z = false;
    n = (array of |int| is 1,2,3,42);
    p = (new |int| is 42);
    free p;
    x = (3 + 4) + 5 * 6 / 7 % 8 + -9 + +10 ^ 2;
    z = z or false and not z;
    if (x == 42) then x = 10; fi
    if (x != 42) then x = 10; fi
    if (x <= 42) then x = 10; fi
    if (x >= 42) then x = 10; fi
    if (x < 42) then x = 10; fi
    if (x > 42) then x = 10; fi
    p = @x;
    *p = 45;
    // y = ++x;
    // y = x++;
    // y = --x;
    // y = x--;
    write "About to write f(x): ";
    write f(x);
    f(x);
    write " Done.\n";
    write "About to write n[3]: ";
    write n[3];
    write " Done.\n";
    write "Please type a new value for n[3]: ";
    read n[3];
    if z then 
        write x; 
    else 
        write y; 
    fi
    while z do
        f(x);
    od
    do
        f(x);
    until z od
    return;
}

function main( ) returns |int|
{
    |string| main_s;
    |bool| main_z;

    test_syntax(main_s, x, y, main_z);
    return 0;
}
