function main( ) returns |int|
{
    |int| x is initially 10;
    |int| y;
    pointer to |int| p;

    y = x++;    // x=11 y=10
    write x; write " "; write y; write "\n";

    y = x--;    // x=10 y=11
    write x; write " "; write y; write "\n";

    y = ++x;    // x=11 y=11
    write x; write " "; write y; write "\n";

    y = --x;    // x=10 y=10
    write x; write " "; write y; write "\n";

    p = @(--x); // x=9 and p points to x
    write x; write " "; write *p; write "\n";

    p = @(++x); // x=10 and p points to x
    write x; write " "; write *p; write "\n";
}

